#include "hip/hip_runtime.h"
// CUDA 2D advection solver test program

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h> //getopt()
#include <assert.h>
#include <sys/time.h> //gettimeofday()
#include <string>   //std::string

#include "serAdvect.h"
#include "parAdvect.h"

#define USAGE   "testAdvect [-h] [-s] [-g Gx[,Gy]] [-b Bx[,By]] [-o] [-w w] [-v v] [-d d] M N [r]"
#define DEFAULTS "Gx=Gy=Bx=By=r=1 v=w=d=0"
#define OPTCHARS "hsg:b:ow:v:d:"

static int M, N;                 // advection field size
static int Gx=1, Gy=1;           // grid dimensions
static int Bx=1, By=1;           // (thread) block dimensions
static int n_timesteps = 1;      // number of timesteps for the simulation
static int opt_host = 0;         // set if -h specified
static int opt_serial = 0;       // set if -s specified
static int opt_optimization = 0; // set if -o specified
static int verbosity = 0;        // v, above
static int w = 0;                // optional extra tuning parameter
static int device_num = 0;       // d, above. id of GPU to be used

// print a usage message for this program and exit with a status of 1
void show_usage_message(std::string msg) {
  printf("testAdvect: %s\n", msg.c_str());
  printf("usage: %s\n\tdefault values: %s\n", USAGE, DEFAULTS);
  fflush(stdout);
  exit(1);
}

void parse_command_line_arguments(int argc, char *argv[]) {
  extern char *optarg; // points to option argument (for -p option)
  extern int optind;   // index of last option parsed by getopt()
  extern int opterr;
  int optchar;        // option character returned my getopt()
  int optD = 0;
  opterr = 0;          // suppress getopt() error message for invalid option
  while ((optchar = getopt(argc, argv, OPTCHARS)) != -1) {
    // extract next option from the command line     
    switch (optchar) {
    case 'h':
      opt_host = 1;
      break;
    case 's':
      opt_serial = 1;
      break;
    case 'g':
      if (sscanf(optarg, "%d,%d", &Gx, &Gy) < 1) // invalid integer 
	      show_usage_message("bad value for Gx");
      break;
    case 'b':
      if (sscanf(optarg, "%d,%d", &Bx, &By) < 1) // invalid integer 
	      show_usage_message("bad value for Bx");
      break;
    case 'o':
      opt_optimization = 1;
      break;
    case 'w':
      if (sscanf(optarg, "%d", &w) != 1) // invalid integer 
	      show_usage_message("bad value for w");
      break;
    case 'v':
      if (sscanf(optarg, "%d", &verbosity) != 1) // invalid integer 
	      show_usage_message("bad value for v");
      break;
    case 'd':
      if (sscanf(optarg, "%d", &device_num) != 1) // invalid integer 
	      show_usage_message("bad value for d");
      optD = 1;
      break;
    default:
      show_usage_message("unknown option");
      break;
    } //switch 
   } //while

  if (optind < argc) {
    if (sscanf(argv[optind], "%d", &M) != 1) 
      show_usage_message("bad value for M");
  } else
    show_usage_message("missing M");
  N = M;
  if (optind+1 < argc)
    if (sscanf(argv[optind+1], "%d", &N) != 1) 
      show_usage_message("bad value for N");
  if (optind+2 < argc)
    if (sscanf(argv[optind+2], "%d", &n_timesteps) != 1) 
      show_usage_message("bad value for r");

  if (opt_host) //ignore -d
    device_num = 0;
  int maxDevices;
  HANDLE_ERROR( hipGetDeviceCount(&maxDevices) );
  if (device_num < 0 || device_num >= maxDevices) {
    printf("warning: device id %d must be in range 0..%d. Using device 0.\n", 
	   device_num, maxDevices-1);
    device_num = 0;
  }
  if (optD)
    HANDLE_ERROR( hipSetDevice(device_num) );
  HANDLE_ERROR( hipGetDevice(&device_num) );

  hipDeviceProp_t prop;
  HANDLE_ERROR( hipGetDeviceProperties( &prop, device_num) );
  if (prop.maxThreadsPerBlock < Bx * By)
    printf("WARNING: Bx=%d By=%d too large for max threads per block = %d %s",
	   Bx, By, prop.maxThreadsPerBlock, "(EXPECT RUBBISH RESULTS)\n"); 
} //getArgs()


static void print_average(std::string name, double total, int nVals) {
  printf("%s %.3e\n", name.c_str(), total / nVals);
}

//return wall time in seconds
static double Wtime() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return(1.0*tv.tv_sec + 1.0e-6*tv.tv_usec);
}

int main(int argc, char** argv) {
  double *u, *u_d = NULL; int ldu, uSize; //advection field
  double t, gflops, t_hd, t_dh; //times

  parse_command_line_arguments(argc, argv);

  printf("Advection of a %dx%d global field on %s %d" 
	 " for %d steps.\n", M, N, opt_host? "host": "GPU", device_num, n_timesteps);
  if (opt_serial)
    printf("\tusing serial computation\n");
  else if (opt_optimization)
    printf("\tusing optimizations (Gx,Gy=%d,%d Bx,By=%d,%d w=%d)\n", 
	   Gx, Gy, Bx, By, w);
  else if (!opt_host)
    printf("\tusing %dx%d blocks of %dx%d threads (2D decomposition)\n", 
	   Gx, Gy, Bx, By);  
  init_advection_parameters(M, N);  
  init_parallel_parameter_values(M, N, Gx, Gy, Bx, By, verbosity);

  ldu = N+2; uSize = (M+2)*ldu*sizeof(double); 
  u = (double *) calloc((M+2)*ldu, sizeof(double)); assert (u != NULL);

  init_advection_field(M, N, &u[ldu + 1], ldu);
  if (verbosity > 1)
    print_advection_field("init u", M, N, &u[ldu + 1], ldu);

  if (!opt_host) {
    HANDLE_ERROR( hipMalloc(&u_d, uSize) );
    t_hd = Wtime();
    HANDLE_ERROR( hipMemcpy(u_d, u, uSize, hipMemcpyHostToDevice) );
    t_hd = Wtime() - t_hd;
  } 
    
  t = Wtime();
  if (opt_host)
    run_serial_advection_host(M, N, n_timesteps, u, ldu);
  else if (opt_serial)
    run_serial_advection_device(M, N, n_timesteps, u_d, ldu);
  else if (opt_optimization)    
    run_parallel_cuda_advection_optimized(n_timesteps, u_d, ldu, w); 
  else
    run_parallel_cuda_advection_2D_decomposition(n_timesteps, u_d, ldu);
  HANDLE_ERROR( hipDeviceSynchronize() );
  t = Wtime() - t;

  gflops = 1.0e-09 * advection_flops_per_element * M * N * n_timesteps;
  printf("Advection time %.2es, GFLOPs rate=%.2e\n", t, gflops / t); 

  if (!opt_host) {
    t_dh = Wtime();
    HANDLE_ERROR( hipMemcpy(u, u_d, uSize, hipMemcpyDeviceToHost) );
    t_dh = Wtime() - t_dh;
    HANDLE_ERROR( hipFree(u_d) );
    printf("Copy times: host-device %.2es, device-host %.2es\n", t_hd, t_dh);
  }
  
  if (verbosity > 1)
    print_advection_field("final u", M+2, N+2, u, ldu);
    print_average("Avg error of final field: ", 
	      compute_error_advection_field(n_timesteps, M, N, &u[ldu + 1], ldu), M*N);
    print_average("Max error of final field: ", 
	      compute_max_error_advection_field(n_timesteps, M, N, &u[ldu + 1], ldu), 1);

  free(u);

  return 0;
} //main()

